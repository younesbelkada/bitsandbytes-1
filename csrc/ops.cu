#include "hip/hip_runtime.h"
#include <ops.cuh>
#include <kernels.cuh>
#include <cub/device/device_scan.cuh>
#include <limits>
#include <cutlass/numeric_types.h>
#include <cutlass/gemm/device/gemm.h>
#include <cassert>

using std::cout;
using std::endl;

template <typename T> void estimateQuantiles(T *A, float *code, float offset, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(code, 0, 256*sizeof(float)));
  kEstimateQuantiles<T><<<blocks, 512>>>(A, code, offset, std::numeric_limits<T>::max(), n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void quantize(float *code, float *A, unsigned char *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kQuantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void dequantize(float *code, unsigned char *A, float *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kDequantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int STOCHASTIC> void quantizeBlockwise(float * code, T *A, float *absmax, unsigned char *out, float *rand, int rand_offset, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
  kQuantizeBlockwise<T, 4096, 4, STOCHASTIC><<<blocks, 1024>>>(code, A, absmax, out, rand, rand_offset, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T> void dequantizeBlockwise(float *code, unsigned char *A, float *absmax, T *out, int blocksize, const int n)
{
  int blocks = n/blocksize;
  blocks = n % blocksize == 0 ? blocks : blocks + 1;
  if(blocksize == 4096)
    kDequantizeBlockwise<T, 4096, 1024, 4><<<blocks, 4096/4>>>(code, A, absmax, out, n);
  else if(blocksize == 2048)
    kDequantizeBlockwise<T, 2048, 512, 4><<<blocks, 2048/4>>>(code, A, absmax, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T, int OPTIMIZER> void optimizer32bit(T* g, T* p, 
                float* state1, float* state2, float *unorm, float max_unorm, float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	switch(OPTIMIZER)
	{
		case ADAM:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit2State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, state2, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
			kOptimizer32bit2State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		case MOMENTUM:
    case RMSPROP:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
				kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, unorm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
			}

			kOptimizer32bit1State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
	}
}

template<typename T, int OPTIMIZER> void optimizerStatic8bit(T* p, T* g,
                unsigned char* state1, unsigned char* state2,
                float *unorm, float max_unorm, float param_norm,
                float beta1, float beta2,
                float eps, int step, float lr, 
                float* quantiles1, float* quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;

  if(max_unorm > 0.0f){ CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float))); }

	switch(OPTIMIZER)
	{
		case ADAM:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			CUDA_CHECK_RETURN(hipMemset(new_max2, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, state2, unorm, beta1, beta2, eps, step, quantiles1, quantiles2, max1, max2, new_max1, new_max2, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, max1, max2, new_max1, new_max2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
    case RMSPROP:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, unorm, beta1, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, eps, step, lr,
																														quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		default:
			break;
	}
}

#define BLOCKSIZE_2STATE 2048
#define NUM_2STATE 8
#define BLOCKSIZE_1STATE 2048
#define NUM_1STATE 8

template<typename T, int OPTIMIZER> void optimizerStatic8bitBlockwise(T* p, T* g,
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr, 
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, int n)
{

	int blocks = 0;
	switch(OPTIMIZER)
	{
		case ADAM:
			blocks = n/BLOCKSIZE_2STATE;
			blocks = n % BLOCKSIZE_2STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit2StateBlockwise<T, OPTIMIZER, BLOCKSIZE_2STATE, NUM_2STATE><<<blocks, BLOCKSIZE_2STATE/NUM_2STATE>>>(p, g, state1, state2, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, absmax1, absmax2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
		case RMSPROP:
			blocks = n/BLOCKSIZE_1STATE;
			blocks = n % BLOCKSIZE_1STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit1StateBlockwise<T, OPTIMIZER, BLOCKSIZE_1STATE, NUM_1STATE><<<blocks, BLOCKSIZE_1STATE/NUM_1STATE>>>(p, g, state1, beta1, beta2, eps, step, lr,
																														quantiles1, absmax1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
	}
}



template<typename T> void percentileClipping(T * g, float *gnorm_vec, int step, const int n)
{
  int blocks = n/2048;
  blocks = n % 2048 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(&gnorm_vec[step % 100], 0, 1*sizeof(float)));
  kPercentileClipping<T, 2048, 4><<<blocks, 512>>>(g, gnorm_vec, step, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

#define UNSIGNED_CHAR 0

void gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

			status = hipblasGemmEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, B, HIP_R_8I, ldb, beta,
					C, HIP_R_32I, ldc,
          HIP_R_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

void strided_gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc, 
                    long long int strideA, long long int strideB, long long int strideC, int batchCount)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

  //cout << transposeA << transposeB << endl;
  //printf("%i %i %i\n", m,n,k);
  //printf("%i %i %i\n", lda,ldb,ldc);
  //printf("%i %i %i\n", strideA, strideB, strideC);
  //printf("%i\n", batchCount);

			status = hipblasGemmStridedBatchedEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, (long long int)strideA, B, HIP_R_8I, ldb, (long long int)strideB, beta,
					C, HIP_R_32I, ldc, (long long int)strideC, batchCount,
          HIP_R_32I, HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}


template<int ORDER> hipblasLtOrder_t get_order()
{
	switch(ORDER)
	{
		case ROW:
      return HIPBLASLT_ORDER_ROW;
			break;
    case COL:
      return HIPBLASLT_ORDER_COL;
      break;
    case COL32:
      return CUBLASLT_ORDER_COL32;
      break;
    case COL_TURING:
      return CUBLASLT_ORDER_COL4_4R2_8C;
      break;
    case COL_AMPERE:
      return CUBLASLT_ORDER_COL32_2R_4R4;
      break;
  }
}

template hipblasLtOrder_t get_order<ROW>();
template hipblasLtOrder_t get_order<COL>();
template hipblasLtOrder_t get_order<COL32>();
template hipblasLtOrder_t get_order<COL_TURING>();
template hipblasLtOrder_t get_order<COL_AMPERE>();


template<int ORDER> int get_leading_dim(int dim1, int dim2)
{
	switch(ORDER)
	{
		case ROW:
      return dim2;
			break;
    case COL:
      return dim1;
      break;
    case COL32:
      // 32*row tiles
      return dim1*32;
      break;
    case COL_TURING:
      return 32*roundoff(dim1, 8);
      break;
    case COL_AMPERE:
      // 32*32 tiles
      return 32*roundoff(dim1, 32);
      break;
  }
}

template int get_leading_dim<ROW>(int dim1, int dim2);
template int get_leading_dim<COL>(int dim1, int dim2);
template int get_leading_dim<COL32>(int dim1, int dim2);

template <typename T, int SRC, int TARGET, bool transpose, int DTYPE> void transform(hipblasLtHandle_t ltHandle, T *A, T *out, int dim1, int dim2)
{

  hipblasLtOrder_t orderA = get_order<SRC>();
  hipblasLtOrder_t orderOut = get_order<TARGET>();
  int ldA = get_leading_dim<SRC>(dim1, dim2);
  int ldOut = get_leading_dim<TARGET>(dim1, dim2);
  
  hipblasLtMatrixLayout_t A_desc = NULL, out_desc = NULL;
  hipblasLtMatrixTransformDesc_t A2Out_desc = NULL;
  hipblasOperation_t opTranspose = HIPBLAS_OP_T;
  float transformAlpha = 1.0f, transformBeta = 0.0f;


  if(DTYPE == 8)
  {
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_8I, dim1, dim2, ldA));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&out_desc, HIP_R_8I, dim1, dim2, ldOut));
  }
  else if(DTYPE == 32)
  {
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_32I, dim1, dim2, ldA));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&out_desc, HIP_R_32I, dim1, dim2, ldOut));
  }
  else
  {
    printf("ERROR WRONG TYPE FOR TRANSFORM: %i\n", DTYPE);
  }

  checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(A_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &orderA, sizeof(orderA)));
  checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(out_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &orderOut, sizeof(orderOut)));

  checkCublasStatus(hipblasLtMatrixTransformDescCreate(&A2Out_desc, HIP_R_32F));

  if(transpose){ checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(A2Out_desc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose))); }

  checkCublasStatus(hipblasLtMatrixTransform(ltHandle, A2Out_desc, &transformAlpha, A, A_desc, &transformBeta, NULL, NULL, out, out_desc, 0));

  if (A_desc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(A_desc));
  if (out_desc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(out_desc));
  if (A2Out_desc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(A2Out_desc));
}

template void transform<int8_t, ROW, COL, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL32, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, ROW, COL32, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_TURING, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_AMPERE, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, COL32, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, COL32, ROW, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);

template <int FORMATB, int DTYPE_OUT, int SCALE_ROWS> int igemmlt(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc) 
{
    int has_error = 0;
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasOperation_t opT = HIPBLAS_OP_T;
    hipblasLtPointerMode_t alphaVec = CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;
    hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t col_turing = CUBLASLT_ORDER_COL4_4R2_8C;
    hipblasLtOrder_t col_ampere = CUBLASLT_ORDER_COL32_2R_4R4;

    has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda));
    has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, n, k, ldb));

    has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
    if(FORMATB == COL_TURING)
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col_turing, sizeof(col_turing)));
    else
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col_ampere, sizeof(col_ampere)));

    if(DTYPE_OUT == 32)
    {
      has_error |= checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
      has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opT, sizeof(opT)));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
      int alpha = 1, beta = 0;
      has_error |= checkCublasStatus(hipblasLtMatmul(ltHandle, matmulDesc,&alpha, A, Adesc, B, Bdesc, &beta, (int32_t*)C, Cdesc, (int32_t*)C, Cdesc, NULL, NULL, 0, 0));
    }
    else
    {
      has_error |= checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32F));
      has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opT, sizeof(opT)));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_8I, m, n, ldc));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
      if(!SCALE_ROWS)
      {
        float alpha = 1.0f, beta = 0.0f;
        has_error |= checkCublasStatus(hipblasLtMatmul(ltHandle, matmulDesc,&alpha, A, Adesc, B, Bdesc, &beta, (int8_t*)C, Cdesc, (int8_t*)C, Cdesc, NULL, NULL, 0, 0));
      }
      else
      {
        has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &alphaVec, sizeof(alphaVec)));
        has_error |= checkCublasStatus(hipblasLtMatmul(ltHandle, matmulDesc, row_scale, A, Adesc, B, Bdesc, NULL, (int8_t*)C, Cdesc, (int8_t*)C, Cdesc, NULL, NULL, 0, 0));
      }
    }


    if (Cdesc) has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) has_error |= checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if(has_error == 1)
      printf("error detected");

    return has_error;
}

void cutlass_igemm(bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{

  // Define the GEMM operation
  using Gemm = cutlass::gemm::device::Gemm<
    int8_t,
    cutlass::layout::ColumnMajor,              // LayoutA
    int8_t,
    cutlass::layout::ColumnMajor,              // LayoutB
    int32_t,
    cutlass::layout::ColumnMajor,              // LayoutOutput
    int32_t,                                     // ElementAccumulator
    cutlass::arch::OpClassWmmaTensorOp,            // tag indicating Tensor Cores
    cutlass::arch::Sm75,                        // tag indicating target GPU compute architecture
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 16, 16>
    //cutlass::gemm::GemmShape<32, 32, 16>
  >;

  //using Gemm = cutlass::gemm::device::Gemm<
  //  int8_t,
  //  cutlass::layout::RowMajor,              // LayoutA
  //  int8_t,
  //  cutlass::layout::ColumnMajor,              // LayoutB
  //  int32_t,
  //  cutlass::layout::ColumnMajor,              // LayoutOutput
  //  int32_t,                                     // ElementAccumulator
  //  cutlass::arch::OpClassTensorOp,            // tag indicating Tensor Cores
  //  cutlass::arch::Sm75                        // tag indicating target GPU compute architecture
  //  //cutlass::gemm::GemmShape<64, 128, 64>,
  //  //cutlass::gemm::GemmShape<32, 32, 64>,
  //  //cutlass::gemm::GemmShape<16, 16, 16>
  //  //cutlass::gemm::GemmShape<32, 32, 16>
  //>;

  Gemm gemm_op;
  cutlass::Status status;

  int alpha = 1;
  int beta = 0;

  int8_t const *ptrA = (int8_t*)A;
  int8_t const *ptrB = (int8_t*)B;
  int32_t const *ptrC = (int32_t*)C;

  int32_t       *ptrD = (int32_t*)C;
	int ldd = ldc;

  //
  // Launch GEMM on the device
  //
  status = gemm_op({
    {m, n, k},
    {ptrA, lda},            // TensorRef to A device tensor
    {ptrB, ldb},            // TensorRef to B device tensor
    {ptrC, ldc},            // TensorRef to C device tensor
    {ptrD, ldd},            // TensorRef to D device tensor - may be the same as C
    {alpha, beta}           // epilogue operation arguments
    });

  if (status != cutlass::Status::kSuccess)
	{
		printf("ERROR\n");
  }
} 

int fill_up_to_nearest_multiple(int value, int multiple)
{
  return value + (value % multiple == 0 ? 0 : (multiple - (value % multiple)));
}

void dequant_mm_int32_fp16(int *A, float *rowStats, float *colStats, half *out, float* newRowStats, float* newcolStats, int numRows, int numCols)
{
  int threads = 512;
  int tileCols = fill_up_to_nearest_multiple(numCols, 32);
  int n = numRows*tileCols;
  int subtile_rows = 128;
  int tilesize = 32*subtile_rows;
  int num_blocks = numRows/subtile_rows;
  num_blocks += (numRows % subtile_rows == 0) ? 0 : 1;
  num_blocks = num_blocks*(tileCols/32);
  assert(threads <= tilesize);

  //cout << num_blocks << " blocks" << endl;

  kdequant_mm_int32_fp16<4, 128, 512><<<num_blocks, threads>>>(A, rowStats, colStats, out, newRowStats, newcolStats, numRows, numCols, tileCols, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

#define STATS_THREADS 64
#define STATS_ITEMS 4
#define STATS_ROWS 16
void getColRowStats(half * A, float *rowStats, float *colStats, int *nnz_count_row, float nnz_threshold, int rows, int cols)
{
  int tile_cols = STATS_THREADS*STATS_ITEMS;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, STATS_ROWS);
  int num_blocks = (tiledCols/tile_cols) * (tiledRows/STATS_ROWS);

  if(nnz_threshold == 0.0)
    kgetColRowStats<half, STATS_THREADS, STATS_ITEMS, STATS_ROWS, STATS_THREADS*STATS_ITEMS, 0><<<num_blocks, STATS_THREADS>>>(A, rowStats, colStats, nnz_count_row, nnz_threshold, rows, cols, tiledRows, tiledCols);
  else if(nnz_threshold != 0.0)
    kgetColRowStats<half, STATS_THREADS, STATS_ITEMS, STATS_ROWS, STATS_THREADS*STATS_ITEMS, 1><<<num_blocks, STATS_THREADS>>>(A, rowStats, colStats, nnz_count_row, nnz_threshold, rows, cols, tiledRows, tiledCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());

}

void doubleRowColQuant(half * A, float *rowStats, float *colStats, char *out_col_normed, char *out_row_normed, int *rowidx, int *colidx, half *val, int *nnz_block_ptr, float threshold, int rows, int cols)
{
  int threads = 64;
  int items_per_thread = 4;
  int tile_cols = threads*items_per_thread;
  int tile_rows = 16;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, tile_rows);
  int num_blocks = (tiledCols/tile_cols) * (tiledRows/tile_rows);

  //cout << cols << " " << tiledCols << " " << tiledRows << endl;
  //cout << "num blocks " << num_blocks << endl;

  //cout << A << " " << out_col_normed << endl;
  if(threshold > 0.0f)
    kDoubleRowColQuant<64, 4, 16, 64*4, 1><<<num_blocks, threads>>>(A, rowStats, colStats, out_col_normed, out_row_normed, rowidx, colidx, val, nnz_block_ptr, threshold, rows, cols, tiledCols);
  else
    kDoubleRowColQuant<64, 4, 16, 64*4, 0><<<num_blocks, threads>>>(A, rowStats, colStats, out_col_normed, out_row_normed, rowidx, colidx, val, nnz_block_ptr, threshold, rows, cols, tiledCols);

  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <int FORMAT, int TRANSPOSE> void transformRowToFormat(char * A, char *out, int rows, int cols)
{
  int threads = 256;
  int items_per_thread = 8;
  // we load 128 column values per warp
  int tile_cols = 32*items_per_thread;
  int tile_rows = 32;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, tile_rows);
  int num_blocks = (tiledCols/tile_cols) * (tiledRows/tile_rows);
  int outCols = fill_up_to_nearest_multiple(cols, 32);
  int outRows = fill_up_to_nearest_multiple(rows, 32);
  if(FORMAT == COL_TURING)
  {
    if(TRANSPOSE)
      outRows = fill_up_to_nearest_multiple(cols, 8);
    else
      outRows = fill_up_to_nearest_multiple(rows, 8);
  }
  else if(FORMAT == COL_AMPERE)
  {
    if(TRANSPOSE)
      outRows = fill_up_to_nearest_multiple(cols, 32);
    else
      outRows = fill_up_to_nearest_multiple(rows, 32);
  }
  else
  {
    if(TRANSPOSE)
    {
      outCols = fill_up_to_nearest_multiple(rows, 32);
      outRows = cols;
    }
  }

  //cout << cols << " " << tiledCols << " " << tiledRows <<  " " << outCols << endl;
  //cout << "num blocks " << num_blocks << endl;

  //cout << A << " " << out_col_normed << endl;
  kTransformRowToFormat<256, 8, 32, 32*8, TRANSPOSE, FORMAT><<<num_blocks, threads>>>(A, out, rows, cols, tiledCols, outRows, outCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void spmm_coo(hipsparseHandle_t handle, int *A_rowidx, int *A_colidx, half *A_vals, int A_nnz, int A_rows, int A_cols, int B_cols, int ldb, half *B, int ldc, half* C, bool transposed_B)
{

    hipsparseSpMatDescr_t descA;
    hipsparseDnMatDescr_t descB, descC;

    float alpha = 1.0f;
    float beta = 0.0f;
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    CHECK_CUSPARSE( hipsparseCreateCoo(&descA, A_rows, A_cols, A_nnz,
                                      A_rowidx, A_colidx, A_vals,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) );
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&descC, A_rows, B_cols, ldc, C,
                                        HIP_R_16F, HIPSPARSE_ORDER_ROW) );
    // Create dense matrix B
    if(transposed_B)
    {
      int tmp = A_cols;
      A_cols = B_cols;
      B_cols = tmp;
    }

    CHECK_CUSPARSE( hipsparseCreateDnMat(&descB, A_cols, B_cols, ldb, B,
                                        HIP_R_16F, HIPSPARSE_ORDER_ROW) );
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 transposed_B ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, descA, descB, &beta, descC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) );
    CUDA_CHECK_RETURN( hipMalloc(&dBuffer, bufferSize) );

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 transposed_B ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, descA, descB, &beta, descC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(descA) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(descB) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(descC) );
    CUDA_CHECK_RETURN( hipFree(dBuffer) );
}

template <typename T, int BITS> void spmm_coo_very_sparse_naive(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, T *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB)
{

  kspmm_coo_very_sparse_naive<T, 8, BITS><<<nnz_rows, 256>>>(max_count, max_idx, offset_rowidx, rowidx, colidx, values, B, out, dequant_stats, nnz, rowsA, rowsB, colsB);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template void spmm_coo_very_sparse_naive<half, 16>(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, half *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB);
template void spmm_coo_very_sparse_naive<signed char, 8>(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, signed char *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB);

template int igemmlt<COL_TURING, 32, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_TURING, 8, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_TURING, 8, 1>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_AMPERE, 32, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_AMPERE, 8, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_AMPERE, 8, 1>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);

template void transformRowToFormat<COL32, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL32, 1>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_TURING, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_TURING, 1>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_AMPERE, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_AMPERE, 1>(char * A, char *out, int rows, int cols);

template void estimateQuantiles(half *A, float *code, float offset, int n);
template void estimateQuantiles(float *A, float *code, float offset, int n);

template void quantizeBlockwise<half, 0>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 0>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<half, 1>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 1>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void dequantizeBlockwise<half>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n);
template void dequantizeBlockwise<float>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n);


#define MAKE_optimizer32bit(name, gtype) \
template void optimizer32bit<gtype, name>(gtype* g, gtype* p, \
                float* state1, float* state2, float* unorm, float max_unorm, float param_norm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const int n);

MAKE_optimizer32bit(ADAM, half)
MAKE_optimizer32bit(ADAM, float)
MAKE_optimizer32bit(MOMENTUM, half)
MAKE_optimizer32bit(MOMENTUM, float)
MAKE_optimizer32bit(RMSPROP, half)
MAKE_optimizer32bit(RMSPROP, float)

#define MAKE_optimizerStatic8bit(name, gtype) \
template void optimizerStatic8bit<gtype, name>(gtype* p, gtype* g, unsigned char* state1, unsigned char* state2, \
                float *unorm, float max_unorm, float param_norm, \
                float beta1, float beta2, \
                float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale, int n); \

MAKE_optimizerStatic8bit(ADAM, half)
MAKE_optimizerStatic8bit(ADAM, float)
MAKE_optimizerStatic8bit(MOMENTUM, half)
MAKE_optimizerStatic8bit(MOMENTUM, float)
MAKE_optimizerStatic8bit(RMSPROP, half)
MAKE_optimizerStatic8bit(RMSPROP, float)

#define MAKE_optimizerStatic8bitBlockwise(gtype, optim_name) \
template void optimizerStatic8bitBlockwise<gtype, optim_name>(gtype* p, gtype* g, \
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, int n); \

MAKE_optimizerStatic8bitBlockwise(half, ADAM);
MAKE_optimizerStatic8bitBlockwise(float, ADAM);
MAKE_optimizerStatic8bitBlockwise(half, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(float, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(half, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(float, RMSPROP);

template void percentileClipping(float * g, float *gnorm_vec, int step, const int n);
template void percentileClipping(half * g, float *gnorm_vec, int step, const int n);
